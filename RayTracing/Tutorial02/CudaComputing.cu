#include "hip/hip_runtime.h"
#include "CudaComputing.cuh"
#include "hip/hip_runtime.h"
#include "hip/device_functions.h"
#include ""
#include "math.h"


__device__ bool HasTheBall ;
__global__ void setDev_ball(bool dev_ball){
	HasTheBall = dev_ball;
}
void setTheBall(bool Ball){
	setDev_ball << <1, 1 >> >(Ball);
}

__device__ bool HasTheCube ;
__global__ void setDev_cube(bool dev_cube){
	HasTheCube = dev_cube;
}
void setTheCube(bool cube){
	setDev_cube << <1, 1 >> >(cube);
}

__device__ bool HasTheCy ;
__global__ void setDev_cy(bool dev_cy){
	HasTheCy = dev_cy;
}

void setTheCylinder(bool Cy){
	setDev_cy << <1, 1 >> >(Cy);
}

__device__ bool HasTheMirror ;
__global__ void setDev_mirror(bool dev_mirror){
	HasTheMirror = dev_mirror;
}

void setTheMirror(bool mi){
	setDev_mirror << <1, 1 >> >(mi);
}

__device__ bool HasTheCurve;
__global__ void setDev_curve(bool dev_curv){
	HasTheCurve = dev_curv;
}

void setTheCurve(bool cur){
	setDev_curve << <1, 1 >> >(cur);
}

__device__ bool HasTheShadow ;
__global__ void setDev_shadow(bool dev_sha){
	HasTheShadow = dev_sha;
}

void setTheShadow(bool sha){
	setDev_shadow << <1, 1 >> >(sha);
}

__device__ bool HasTheBallFlection;
__global__ void setDev_BF(bool dev_sha){
	HasTheBallFlection = dev_sha;
}

void setTheBF(bool sha){
	setDev_BF<< <1, 1 >> >(sha);
}



__device__ float CyHeight = 250;

__device__ float CubeX = 600;
__device__ float CubeY = 0;
__device__ float CubeZ = -400;

__device__ float CyX = 800;
__device__ float CyY = 0;
__device__ float CyZ = -300;


__device__ bool chekcSolution(float a, float b, float c){
	if ((b*b - 4 * a*c)<0)return false;
	return true;
}

__device__ float getSolution1(float a, float b, float c){
	float rst = -b + sqrt(b*b - 4 * a*c);
	rst = rst / (2 * a);
	return rst;
}

__device__ float getSolution2(float a, float b, float c){
	float rst = -b - sqrt(b*b - 4 * a*c);
	rst = rst / (2 * a);
	return rst;
}

__device__ float dot(float3 a, float3 b){
	float c;
	c = a.x*b.x + a.y*b.y + a.z*b.z;
	return c;
}

__device__ float3 normalize(float3 n){
	float length1 = n.x*n.x + n.y*n.y + n.z*n.z;
	float length = sqrt(length1);

	n.x = n.x / length;
	n.y = n.y / length;
	n.z = n.z / length;
	return n;
}

__device__ float bigger(float a, float b){
	if (a > b)return a;
	return b;
}


__device__ bool IsHitTheCube(float3 s, float3 center, float e){

	float up = center.y + e;
	float down = center.y;
	float left = center.x - e / 2;
	float right = center.x + e / 2;
	float front = center.z + e / 2;
	float back = center.z - e / 2;

	if (s.y <= up&&s.y >= down&&s.x >= left&&s.x <= right&&s.z <= front&&s.z >= back){
		return true;
	}
	return false;

}
//�����ģ��߳�
__device__ float4 HitTheCube(float3 t,float3 d,float3 center,float e){
	float up=center.y+e;
	float down=center.y;
	float left=center.x-e/2;
	float right=center.x+e/2;
	float front=center.z+e/2;
	float back=center.z-e/2;

	if (t.x - d.x * 5 > right&&t.x <= right){
		return(make_float4(t.x - d.x * 2.5, t.y - d.y * 2.5, t.z - d.z * 2.5, 1.0));
	}
	if (t.x - d.x * 5 < left&&t.x >= left){
		return(make_float4(t.x - d.x * 2.5, t.y - d.y * 2.5, t.z - d.z * 2.5, 2.0));
	}
	if (t.y - d.y * 5 > up&&t.y <= up){
		return(make_float4(t.x - d.x * 2.5, t.y - d.y * 2.5, t.z - d.z * 2.5, 3.0));
	}
	if (t.y - d.y * 5 < down&&t.y >= down){
		return(make_float4(t.x - d.x * 2.5, t.y - d.y * 2.5, t.z - d.z * 2.5, 4.0));
	}
	if (t.z - d.z * 5 > front&&t.z <= front){
		return(make_float4(t.x - d.x * 2.5, t.y - d.y * 2.5, t.z - d.z * 2.5, 5.0));
	}
	if (t.z - d.z * 5 < back&&t.z >= back){
		return(make_float4(t.x - d.x * 2.5, t.y - d.y * 2.5, t.z - d.z * 2.5, 7.0));
	}

	return make_float4(0.0, 0.0, 0.0, 0.0);
}

__device__ bool IsHitTheCylinder(float3 s,float3 c,float r,float h){
	if ((s.x - c.x)*(s.x - c.x) + (s.z - c.z)*(s.z - c.z) <= r*r&&s.y <= h&&s.y>=0){
		return true;
	}
	return false;
}
//�����ģ��뾶���߶�
__device__ float4 HitTheCylinder(float3 t,float3 d,float3 c,float r,float h){
	if(t.y <= h&&t.y - d.y * 5>h){
		return make_float4(t.x, t.y, t.z, 3.0);
	}
	if ((t.x - c.x)*(t.x - c.x) + (t.z - c.z)*(t.z - c.z) <= r*r &&
		(t.x - d.x * 5 - c.x)*(t.x - d.x * 5 - c.x) + (t.z - d.z * 5 - c.z)*(t.z - d.z * 5 - c.z) > r*r){
		return make_float4(t.x, t.y, t.z, 9.0);
	}
}


__device__ float4 rayFromShpere(float3 s, float3 dir){
	float4 rst;
	rst.x = 0.0;
	rst.y = 0.0;
	rst.z = 0.0;
	rst.w = 7.0;


	float k;

	float x;
	float y;
	float z;

	float3 d = normalize(dir);


	float R = 140;

	float3 t = s;

	for (int i = 0; i < 100; i++){
		t.x += d.x * 5;
		t.y += d.y * 5;
		t.z += d.z * 5;

		if (HasTheCube&&IsHitTheCube(t, make_float3(CubeX,CubeY,CubeZ), 200)){
			return HitTheCube(t, d, make_float3(CubeX, CubeY, CubeZ), 200);
		}
		if (HasTheCy&&IsHitTheCylinder(t, make_float3(CyX, CyY, CyZ), 100, CyHeight)){
			return HitTheCylinder(t, d, make_float3(CyX, CyY, CyZ), 100, CyHeight);
		}
		//z = 0; 7.0
		if (t.z >= 0 && t.z - 5 * d.z < 0){
			rst.x = t.x - d.x*2.5;
			rst.y = t.y - d.y*2.5;
			rst.z = t.z - d.z*2.5;
			rst.w = 7.0;
			return rst;
		}

		//z=-600 5.0

		//x = 0; 1.0
		if (t.x <= 0 && t.x - 5 * d.x > 0){
			rst.x = t.x - d.x*2.5;
			rst.y = t.y - d.y*2.5;
			rst.z = t.z - d.z*2.5;
			rst.w = 1.0;
			return rst;
		}


		//x = 1200; 2.0
		if (t.x >= 1200 && t.x - 5 * d.x < 1200){
			rst.x = t.x - d.x*2.5;
			rst.y = t.y - d.y*2.5;
			rst.z = t.z - d.z*2.5;
			rst.w = 2.0;
			return rst;
		}


		//y = 0;  3.0
		if (t.y <= 0 && t.y - 5 * d.y > 0){
			rst.x = t.x - d.x*2.5;
			rst.y = t.y - d.y*2.5;
			rst.z = t.z - d.z*2.5;
			rst.w = 3.0;
			return rst;
		}


		//y = 600;  4.0
		if (t.y >= 600 && t.y - 5 * d.y < 600){
			rst.x = t.x - d.x*2.5;
			rst.y = t.y - d.y*2.5;
			rst.z = t.z - d.z*2.5;
			rst.w = 4.0;
			return rst;
		}


	}

	return rst;
}

__device__ bool IsHitTheBall(float3 e, float3 p, float3 cen, float R){
	float a = (p.x - e.x)*(p.x - e.x) + (p.y - e.y)*(p.y - e.y) + (p.z - e.z)*(p.z - e.z);
	float b = 2 * ((p.x - e.x)*(e.x - cen.x) + (p.y - e.y)*(e.y - cen.y) + (p.z - e.z)*(e.z - cen.z));
	float c = (e.x - cen.x)*(e.x - cen.x) + (e.y - cen.y)*(e.y - cen.y) + (e.z - cen.z)*(e.z - cen.z) - R*R;
	if (chekcSolution(a, b, c) == true){
		return true;
	}
	return false;
}

__device__ float4 HitTheBall(float3 e, float3 p,float3 cen,float R){
	float4 rst;
	rst.x = 0.0;
	rst.y = 0.0;
	rst.z = 0.0;
	rst.w = 0.0;

	float k;

	float a = (p.x - e.x)*(p.x - e.x) + (p.y - e.y)*(p.y - e.y) + (p.z - e.z)*(p.z - e.z);
	float b = 2 * ((p.x - e.x)*(e.x - cen.x) + (p.y - e.y)*(e.y - cen.y) + (p.z - e.z)*(e.z - cen.z));
	float c = (e.x - cen.x)*(e.x - cen.x) + (e.y - cen.y)*(e.y - cen.y) + (e.z - cen.z)*(e.z - cen.z) - R*R;

	//hit the ball
	k = getSolution1(a, b, c);
	rst.x = (p.x - e.x)*k + e.x;
	rst.y = (p.y - e.y)*k + e.y;
	rst.z = (p.z - e.z)*k + e.z;
	rst.w = 6.0;
	float3 L1 = make_float3((p.x - rst.x), (p.y - rst.y), (p.z - rst.z));
	L1 = normalize(L1);
	float3 N = make_float3((rst.x - cen.x), (rst.y - cen.y), (rst.z - cen.z));
	N = normalize(N);
	float3 L2 = make_float3(-2 * dot(L1, N)*N.x + L1.x, -2 * dot(L1, N)*N.y + L1.y, -2 * dot(L1, N)*N.z + L1.z);
	//����ѡ��
	if (HasTheBallFlection)return rayFromShpere(make_float3(rst.x, rst.y, rst.z), L2);
	return rst;
	
}

__device__ float4 HitTheMirror(float3 e, float3 p, float3 cen){
	float4 rst;
	rst.x = 0.0;
	rst.y = 0.0;
	rst.z = 0.0;
	rst.w = 7.0;


	float k;

	float x;
	float y;
	float z;



	float R = 140;

	if (HasTheBall&&IsHitTheBall(e, p, cen, R) == true){
		return HitTheBall(e, p, cen, R);
	}

	float3 d = normalize(make_float3(p.x - e.x, p.y - e.y, p.z - e.z));
	float3 t = p;
	for (int i = 0; i < 200; i++){
		t = make_float3(t.x + d.x * 5, t.y + d.y * 5, t.z + d.z * 5);
		if (HasTheCube&&IsHitTheCube(t, make_float3(CubeX, CubeY, CubeZ), 200)){
			return HitTheCube(t, d, make_float3(CubeX, CubeY, CubeZ), 200);
		}
		if (HasTheCy&&IsHitTheCylinder(t, make_float3(CyX, CyY, CyZ), 100, CyHeight)){
			return HitTheCylinder(t, d, make_float3(CyX, CyY, CyZ), 100, CyHeight);
		}
	}

	z = 0;
	k = (z - e.z) / (p.z - e.z);
	x = (p.x - e.x)*k + e.x;
	y = (p.y - e.y)*k + e.y;
	if (x >= 0 && x <= 1200 && y >= 0 && y <= 600){
		rst.x = x;
		rst.y = y;
		rst.z = z;
		rst.w = 7.0;

		return rst;
	}

	x = 0;
	k = (x - e.x) / (p.x - e.x);
	y = (p.y - e.y)*k + e.y;
	z = (p.z - e.z)*k + e.z;
	if (y >= 0 && y <= 600 && z >= -600 && z <= 0){
		rst.x = x;
		rst.y = y;
		rst.z = z;
		rst.w = 1.0;
		return rst;
	}

	x = 1200;
	k = (x - e.x) / (p.x - e.x);
	y = (p.y - e.y)*k + e.y;
	z = (p.z - e.z)*k + e.z;
	if (y >= 0 && y <= 600 && z >= -600 && z <= 0){
		rst.x = x;
		rst.y = y;
		rst.z = z;
		rst.w = 2.0;
		return rst;
	}

	y = 0;
	k = (y - e.y) / (p.y - e.y);
	x = (p.x - e.x)*k + e.x;
	z = (p.z - e.z)*k + e.z;
	if (x >= 0 && x <= 1200 && z >= -600 && z <= 0){
		rst.x = x;
		rst.y = y;
		rst.z = z;
		rst.w = 3.0;
		return rst;
	}

	y = 600;
	k = (y - e.y) / (p.y - e.y);
	x = (p.x - e.x)*k + e.x;
	z = (p.z - e.z)*k + e.z;
	if (x >= 0 && x <= 1200 && z >= -600 && z <= 0){
		rst.x = x;
		rst.y = y;
		rst.z = z;
		rst.w = 4.0;
		return rst;
	}


	return rst;
}

__device__ float4 HitCurveMirror(float3 s, float3 d,float3 ball){
	float4 rst;
	rst.x = 0.0;
	rst.y = 0.0;
	rst.z = 0.0;
	rst.w = 7.0;


	float3 L1;
	float3 N;
	float3 L2;

	float3 t = s;
	d = normalize(d);
	//hit poin
	for (int i = 0; i < 200; i++){
		t.x += d.x * 5;
		t.y += d.y * 5;
		t.z += d.z * 5;
		if (t.y>500)return(make_float4(t.x, t.y, t.z, 4.0));
		if ((t.x - 600)*(t.x - 600) + (t.z + 225)*(t.z + 225) >= 625 * 625 && (t.x - d.x * 5 - 600)*(t.x - d.x * 5 - 600) + (t.z - d.z * 5 + 225)*(t.z - d.z * 5 + 225) < 625 * 625){
			L1 = make_float3(-d.x, -d.y, -d.z);
			L1 = normalize(L1);
			N = make_float3(600 - t.x, 0,-225 - t.z);
			N = normalize(N);
			L2 = make_float3(2 * dot(L1, N)*N.x - L1.x, 2 * dot(L1, N)*N.y - L1.y, 2 * dot(L1, N)*N.z - L1.z);
			
			return HitTheMirror(t, make_float3(t.x + L2.x, t.y + L2.y, t.z + L2.z), ball);
			break;
		}
	}


	return rst;
}

__device__ float4 HitTheWall(float3 e,float3 p,float3 cen){
	float4 rst;
	rst.x = 0.0;
	rst.y = 0.0;
	rst.z = 0.0;
	rst.w = 0.0;

	float x;
	float y;
	float z;

	float k;

	z = -600;
	k = (z - e.z) / (p.z - e.z);
	x = (p.x - e.x)*k + e.x;
	y = (p.y - e.y)*k + e.y;
	if (x >= 0 && x <= 1200 && y >= 0 && y <= 600){
		
		if (x >= 100 && x <= 1100 && y >= 100 && y <= 550){
			if (HasTheMirror){
				if (HasTheCurve){
					return HitCurveMirror(make_float3(p.x, p.y, p.z), make_float3(p.x - e.x, p.y - e.y, p.z - e.z), make_float3(cen.x, cen.y, cen.z));
				}
				return HitTheMirror(make_float3(e.x,e.y,-1200-e.z), make_float3(x,y,z), cen);
			}
			if (!HasTheMirror){
				rst.x = x;
				rst.y = y;
				rst.z = z;
				rst.w = 5.0;
				return rst;
			}
			
		}
		else{
			rst.x = x;
			rst.y = y;
			rst.z = z;
			rst.w = 5.0;
			return rst;
		}
	}

	x = 0;
	k = (x - e.x) / (p.x - e.x);
	y = (p.y - e.y)*k + e.y;
	z = (p.z - e.z)*k + e.z;
	if (y >= 0 && y <= 600 && z >= -600 && z <= 0){
		rst.x = x;
		rst.y = y;
		rst.z = z;
		rst.w = 1.0;
		return rst;
	}

	x = 1200;
	k = (x - e.x) / (p.x - e.x);
	y = (p.y - e.y)*k + e.y;
	z = (p.z - e.z)*k + e.z;
	if (y >= 0 && y <= 600 && z >= -600 && z <= 0){
		rst.x = x;
		rst.y = y;
		rst.z = z;
		rst.w = 2.0;
		return rst;
	}

	y = 0;
	k = (y - e.y) / (p.y - e.y);
	x = (p.x - e.x)*k + e.x;
	z = (p.z - e.z)*k + e.z;
	if (x >= 0 && x <= 1200 && z >= -600 && z <= 0){
		rst.x = x;
		rst.y = y;
		rst.z = z;
		rst.w = 3.0;
		return rst;
	}

	y = 600;
	k = (y - e.y) / (p.y - e.y);
	x = (p.x - e.x)*k + e.x;
	z = (p.z - e.z)*k + e.z;
	if (x >= 0 && x <= 1200 && z >= -600 && z <= 0){
		rst.x = x;
		rst.y = y;
		rst.z = z;
		rst.w = 4.0;
		if ((x - 600)*(x - 600) + (z + 300)*(z + 300)<100 * 100)rst.w = 8.0;
		return rst;
	}


	return rst;
}

__device__ float4 getHitPoint(float3 e, float3 p, float3 cen){
	
	
    //hit the ball
	float R = 140;

	if (IsHitTheBall(e, p, cen, R) == true && HasTheBall==true){
		return HitTheBall(e, p, cen, R);
	}
	//hit the cube and the cylinder
	float3 d = normalize(make_float3(p.x-e.x,p.y-e.y,p.z-e.z));
	float3 t = p;
	for (int i = 0; i < 100; i++){
		t = make_float3(t.x + d.x * 5, t.y + d.y * 5, t.z + d.z * 5);
		if (HasTheCube&&IsHitTheCube(t, make_float3(CubeX, CubeY, CubeZ), 200)){
			return HitTheCube(t, d, make_float3(CubeX, CubeY, CubeZ), 200);
		}
		if (HasTheCy&&IsHitTheCylinder(t, make_float3(CyX, CyY, CyZ), 100, CyHeight)){
			return HitTheCylinder(t, d, make_float3(CyX, CyY, CyZ), 100, CyHeight);
		}
	}

	

	//hit the wall
	
	return HitTheWall(e, p, cen);
}

__device__ float3 getNormal(float4 p,float cx,float cy,float cz){
	float3 N;
	

	if (p.w != 0.0){
		if (p.w == 6.0){
			N = make_float3(p.x - cx, p.y - cy, p.z - cz);
		}
		if (p.w == 5.0){
			N = make_float3(0, 0, 1);
		}
		if (p.w == 1.0){
			N = make_float3(1, 0, 0);
		}
		if (p.w == 2.0){
			N = make_float3(-1, 0, 0);
		}
		if (p.w == 3.0){
			N = make_float3(0, 1, 0);
		}
		if (p.w == 4.0){
			N = make_float3(0, -1, 0);
		}
		if (p.w == 7.0){
			N = make_float3(0, 0, -1);
		}
		if (p.w == 9.0){
			N = make_float3(p.x-800,0,p.z+300);
		}
	}

		N = normalize(N);
		return N;
}



__device__ float4 getColor(float4 p,float3 n,float ex,float ey,float ez){
	
	


	float dist = (p.x - ex)*(p.x - ex) + (p.y - ey)*(p.y - ey) + (p.z - ez)*(p.z - ez);
	dist /= 1200000;
	if (dist < 1)dist = 1;
	
	//�����cuda dyD,dyS,dyA ���ɲ���
	float4 kd = make_float4(0.5, 0.5, 0.5, 1.0);

	float4 ks = make_float4(0.0, 0.0, 0.1, 1.0);

	float4 ka = make_float4(0.1, 0.1, 0.1, 1.0);


	
	float4 dyDiffuse = make_float4(1.0, 1.0, 1.0, 1.0);

	float4 dySpecular = make_float4(0.5, 0.5, 0.5, 1.0);

    float4 dyAmbient = make_float4(0.2, 0.2, 0.2, 1.0);


	

	if (p.w == 6.0){//the ball
		kd = make_float4(0.5, 0.5, 0.9, 1.0);

		ks = make_float4(0.0, 0.0, 0.0, 1.0);

		ka = make_float4(0.5, 0.5, 0.5, 1.0);
	}
	if (p.w == 5.0){//back wall
		kd = make_float4(0.0, 0.6, 0.0, 1.0);

		ks = make_float4(0.9, 0.0, 0.0, 1.0);

		ka = make_float4(0.05, 0.0, 0.0, 1.0);
	}
	if (p.w == 1.0){//left wall
		kd = make_float4(0.5, 0.0, 0.0, 1.0);

		ks = make_float4(0.1, 0.0, 0.0, 1.0);

		ka = make_float4(0.9, 0.9, 0.1, 1.0);
	}
	if (p.w == 2.0){//right wall
		kd = make_float4(0.0, 0.0, 0.5, 1.0);

		ks = make_float4(0.1, 0.0, 0.0, 1.0);

		ka = make_float4(0.9, 0.9, 0.1, 1.0);
	}
	if (p.w == 3.0){//floor
		kd = make_float4(0.0, 0.5, 0.5, 1.0);

		ks = make_float4(1.0, 1.0, 1.0, 1.0);

		ka = make_float4(0.9, 0.9, 0.1, 1.0);
	}
	if (p.w == 4.0){//ceil
		kd = make_float4(0.0, 0.5, 0.5, 1.0);

		ks = make_float4(1.0, 1.0, 1.0, 1.0);

		ka = make_float4(0.9, 0.9, 0.1, 1.0);
	}
	if (p.w == 7.0){//front wall
		kd = make_float4(0.5, 0.0, 0.7, 1.0);

		ks = make_float4(0.4, 0.4, 0.4, 1.0);

		ka = make_float4(0.4, 0.4, 0.4, 1.0);
	}

	if (p.w == 9.0){
		kd = make_float4(0.0, 1.0, 1.0, 1.0);

		ks = make_float4(0.4, 0.4, 0.4, 1.0);

		ka = make_float4(0.4, 0.4, 0.4, 1.0);
	}
	

	float3 V = normalize(make_float3(ex - p.x, ey - p.y, ez - p.z));


	float3 L = normalize(make_float3(600 - p.x, 600 - p.y, -300 - p.z));

	

	float3 H = normalize(make_float3(V.x + L.x, V.y + L.y, V.z + L.z));

	float4 ambient1 = make_float4(ka.x*dyAmbient.x , ka.y*dyAmbient.y , ka.z*dyAmbient.z , ka.w*dyAmbient.w );
	
	float max1 = bigger(dot(n, L), 0.0f);
	float4 diffuse1 = make_float4(kd.x*max1*dyDiffuse.x / dist, kd.y*max1*dyDiffuse.y / dist, kd.z*max1*dyDiffuse.z / dist, kd.w*max1*dyDiffuse.w / dist);
		
	float max2 = powf(bigger(dot(n, H), 0.0f),10.0f);
	float4 specular1 = make_float4(ks.x*max2*dySpecular.x,ks.y*max2*dySpecular.y, ks.z*max2*dySpecular.z, ks.w*max2*dySpecular.w);
	
	if(dot(n,L)<0) specular1 =make_float4(0.0,0.0,0.0,0.0);






	float4 color1 = make_float4(ambient1.x + diffuse1.x+specular1.x, 
		ambient1.y + diffuse1.y + specular1.y,
		ambient1.z + diffuse1.z + specular1.z,
		ambient1.w + diffuse1.w + specular1.w);

	if (p.w == 8.0){
		color1 = make_float4(1.0, 1.0, 0.0, 1.0);
	}

	return color1;
}

__device__ bool shadowRay(float3 s, float3 e, float3 center, float R){
	int divide = 100;
	float divX = (e.x - s.x) / divide;
	float divY = (e.y - s.y) / divide;
	float divZ = (e.z - s.z) / divide;
	float3 t = s;
	for (int i = 0; i < divide; i++){
		t.x += divX;
		t.y += divY;
		t.z += divZ;
		if (HasTheBall&&((t.x - divX - center.x)*(t.x - divX - center.x) + (t.y - divY - center.y)*(t.y - divY - center.y) + (t.z - divZ - center.z)*(t.z - divZ - center.z) > R*R) && ((t.x - center.x)*(t.x - center.x) + (t.y - center.y)*(t.y - center.y) + (t.z - center.z)*(t.z - center.z) <= R*R)){
			return true;
			break;
		}
		if (HasTheCube&&IsHitTheCube(t, make_float3(CubeX, CubeY, CubeZ), 200) == true){
			return true;
			break;
		}
		if (HasTheCy&&IsHitTheCylinder(t, make_float3(CyX, CyY, CyZ), 100, CyHeight)){
			return true;
			break;
		}
	}
	return false;
}

//global
__global__ void computeSingleRay(char* tex){
	//vec4 temp = getHitPoint(Ex, Ey, Ez, vPosition.x, vPosition.y, vPosition.z);
	//width height Ӧ���ǲ���
	//position=thread.x
	//�̴߳�����λ��
	//int j = threadIdx.x;
	//int i = blockIdx.x;
	
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;

	float Ex = 600;
	float Ey = 300;
	float Ez = 800;
	
	float Cx = 200;
	float Cy = 300;
	float Cz = -350;

	float3 E = make_float3(Ex, Ey, Ez);
	float3 P = make_float3(i, j, 0);
	float3 C = make_float3(Cx,Cy, Cz);
	//float Cx = 200;
	

//�����hit ��λ�� float4
	float4 position = getHitPoint(E,P,C);

//����λ�����normal	
	float3 normal = getNormal(position,Cx,Cy,Cz);
//��normal�����ɫ vec4
	float4 color = getColor(position, normal,Ex,Ey,Ez);
	
	float3 p = make_float3(position.x, position.y, position.z);
	float3 e = make_float3(600, 600, -300);
	float3 c = make_float3(Cx, Cy, Cz);
	
	if (HasTheShadow&&shadowRay(p, e, c, 140) && position.w != 6.0)color = make_float4(color.x*0.2, color.y*0.2, color.z*0.2, 1);

	tex[j * 1200 * 3 + i * 3] = color.x*255;
	tex[j * 1200 * 3 + i * 3 + 1] = color.y*255;
	tex[j * 1200 * 3 + i * 3 + 2] = color.z*255;
}

//1200*600 size, Ex Ey Ez
void computeRays(int width,int height,char *tex){

	char * dev_Tex;

	hipMalloc((char**)&dev_Tex, 3 * width * height * sizeof(char));

	dim3 block(8, 8, 1);
	dim3 grid(width/ block.x, height / block.y, 1);
	
	computeSingleRay << <grid, block >> >(dev_Tex);

	hipMemcpy(tex, dev_Tex, 3 * width * height * sizeof(char), hipMemcpyDeviceToHost);

	hipFree(dev_Tex);
}


